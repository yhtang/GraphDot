#include "hip/hip_runtime.h"
#include <limits>
#include <basekernel.h>
#include <graph.h>
#include <marginalized_kernel.h>
#include <fmath.h>
#include <frozen_array.h>
#include <numpy_type.h>
#include <util_cuda.h>

using namespace graphdot::cuda;
using namespace graphdot::numpy_type;
using namespace graphdot::basekernel;
namespace solver_ns = graphdot::marginalized;

${node_kernel}
${edge_kernel}
${p_start}

using node_t = ${node_t};
using edge_t = ${edge_t};

using graph_t   = graphdot::graph_t<node_t, edge_t>;
using scratch_t = solver_ns::pcg_scratch_t;
using solver_t  = solver_ns::labeled_compact_block_dynsched_pcg<graph_t>;

__constant__ char shmem_bytes_per_warp[solver_t::shmem_bytes_per_warp];

extern "C" {
    __global__ void graph_kernel_solver(
        graph_t const   * graphs,
        float32        ** diags,
        scratch_t       * scratches,
        uint2           * jobs,
        uint32          * starts,
        float32         * out,
        uint32          * i_job_global,
        const uint32      n_jobs,
        const uint32      out_h,
        const uint32      out_w,
        const float32     q,
        const float32     q0
    ) {
        extern __shared__ char shmem[];
        __shared__ uint32 i_job;

        auto scratch = scratches[blockIdx.x];

        while (true) {
            if (threadIdx.x == 0) i_job = atomicInc(i_job_global, 0xFFFFFFFF);
            __syncthreads();

            if (i_job >= n_jobs) break;

            auto const job = jobs[i_job];
            auto const g1  = graphs[job.x];
            auto const g2  = graphs[job.y];
            auto const I1  = starts[job.x];
            auto const I2  = starts[job.y];
            const int  n1  = g1.n_node;
            const int  n2  = g2.n_node;
            const int   N  = n1 * n2;
            auto const diag1 = diags[job.x];
            auto const diag2 = diags[job.y];

            if (?{traits.eval_gradient is True}) {
                solver_t::compute_duo(
                    node_kernel,
                    edge_kernel,
                    p_start,
                    g1, g2,
                    scratch,
                    shmem,
                    q, q0);
            } else {
                solver_t::compute(
                    node_kernel,
                    edge_kernel,
                    g1, g2,
                    scratch,
                    shmem,
                    q, q0);
            }
            __syncthreads();

            /********* post-processing *********/

            // apply starting probability and min-path truncation
            if (?{traits.lmin == 1}) {
                for (int i = threadIdx.x; i < N; i += blockDim.x) {
                    int i1 = i / n2;
                    int i2 = i % n2;
                    scratch.x(i) -= node_kernel(g1.node[i1], g2.node[i2]) * q * q / (q0 * q0);
                }
            }

            // reusing r for d12 and z for d21
            auto d12 = scratch.r();
            auto d21 = scratch.z();
            for(int i = threadIdx.x; i < max(n1, n2); i += blockDim.x) {
                d12[i] = std::numeric_limits<float32>::max();
                d21[i] = std::numeric_limits<float32>::max();
            }
            __syncthreads();

            for (int i = threadIdx.x; i < N; i += blockDim.x) {
                int i1 = i / n2;
                int i2 = i % n2;
                auto r12 = scratch.x(i) * p_start(g1.node[i1]) * p_start(g2.node[i2]);
                auto r1 = diag1[i1];
                auto r2 = diag2[i2];
                auto k = r12 * rsqrtf(r1 * r2);
                auto d = sqrtf(2 - 2 * max(k, 0.f));
                atomicMin(d12 + i1, d);
                atomicMin(d21 + i2, d);
            }
            __syncthreads();

            for (int i = threadIdx.x; i < n1; i += blockDim.x) {
                atomicMax(d12, d12[i]);
            }
            for (int i = threadIdx.x; i < n2; i += blockDim.x) {
                atomicMax(d21, d21[i]);
            }
            __syncthreads();

            // write to output buffer
            if (graphdot::cuda::laneid() == 0) {
                auto dh = max(*d12, *d21);
                out[I1 + I2 * out_h] = dh;
                if (?{traits.symmetric is True}) {
                    if (job.x != job.y) {
                        out[I2 + I1 * out_h] = dh;
                    }
                }
            }
            __syncthreads();

            // if (?{traits.eval_gradient is True}) {

            //     constexpr static int jac_starts[] {
            //         0,
            //         p_start.jac_dims,
            //         p_start.jac_dims + 1,
            //         p_start.jac_dims + 1 + node_kernel.jac_dims,
            //         p_start.jac_dims + 1 + node_kernel.jac_dims + edge_kernel.jac_dims
            //     };

            //     __shared__ float jac[jac_starts[4]];

            //     for (int i = threadIdx.x; i < jac_starts[4]; i += blockDim.x) {
            //         jac[i] = 0;
            //     }
            //     __syncthreads();

            //     solver_t::derivative_p(
            //         p_start,
            //         g1, g2,
            //         scratch,
            //         shmem,
            //         jac + jac_starts[0]);

            //     solver_t::derivative_q(
            //         node_kernel,
            //         p_start,
            //         g1, g2,
            //         scratch,
            //         shmem,
            //         jac + jac_starts[1],
            //         q);

            //     solver_t::derivative_node(
            //         node_kernel,
            //         g1, g2,
            //         scratch,
            //         shmem,
            //         jac + jac_starts[2],
            //         q);

            //     solver_t::derivative_edge(
            //         edge_kernel,
            //         g1, g2,
            //         scratch,
            //         shmem,
            //         jac + jac_starts[3]);

            //     for (int i = threadIdx.x; i < jac_starts[4]; i += blockDim.x) {
            //         out[I1 + I2 * out_h + (i + 1) * out_h * out_w] = jac[i];
            //         if (?{traits.symmetric is True}) {
            //             if (job.x != job.y) {
            //                 out[I2 + I1 * out_h + (i + 1) * out_h * out_w] = jac[i];
            //             }
            //         }
            //     }
            // }
        }
    }
}
