#include "hip/hip_runtime.h"
#include <array.h>
#include <basekernel.h>
#include <fmath.h>
#include <frozen_array.h>
#include <graph.h>
#include <marginalized_kernel.h>
#include <numpy_type.h>
#include <tensor_view.h>
#include <util_cuda.h>

using namespace graphdot::numpy_type;
using namespace graphdot::basekernel;
namespace solver_ns = graphdot::marginalized;

${node_kernel}
${edge_kernel}
${p_start}

using node_t = ${node_t};
using edge_t = ${edge_t};

using graph_t   = graphdot::graph_t<node_t, edge_t>;
using scratch_t = solver_ns::pcg_scratch_t;
using solver_t  = solver_ns::labeled_compact_block_dynsched_pcg<graph_t>;

__constant__ char shmem_bytes_per_warp[solver_t::shmem_bytes_per_warp];

extern "C" {
    __global__ void graph_kernel_solver(
        graph_t const   * graphs,
        scratch_t       * scratches,
        uint2           * jobs,
        uint            * starts,
        float32         * gramian,
        float32         * gradient,
        uint            * i_job_global,
        const uint        n_jobs,
        const uint        nX,
        const uint        nY,
        const uint        nJ,
        const float32     q,
        const float32     q0
    ) {
        extern __shared__ char shmem[];
        __shared__ uint i_job;

        const int lane = graphdot::cuda::laneid();
        auto scratch = scratches[blockIdx.x];

        while (true) {
            if (threadIdx.x == 0) i_job = atomicInc(i_job_global, 0xFFFFFFFF);
            __syncthreads();

            if (i_job >= n_jobs) break;

            const auto job = jobs[i_job];
            const auto g1  = graphs[job.x];
            const auto g2  = graphs[job.y];
            const auto I1  = starts[job.x];
            const auto I2  = starts[job.y];
            const uint n1  = g1.n_node;
            const uint n2  = g2.n_node;
            const uint N   = n1 * n2;

            // setup kernel matrix view
            #if ?{traits.nodal == "block"}
                auto K = graphdot::tensor_view(gramian, nX);
            #elif ?{traits.nodal is True}
                #if ?{traits.diagonal is True}
                    auto K = graphdot::tensor_view(gramian, nX);
                #else
                    auto K = graphdot::tensor_view(gramian, nX, nY);
                #endif
            #elif ?{traits.nodal is False}
                #if ?{traits.diagonal is True}
                    auto K = graphdot::tensor_view(gramian, nX);
                #else
                    auto K = graphdot::tensor_view(gramian, nX, nY);
                #endif
            #endif

            // setup Jacobian matrix view
            #if ?{traits.eval_gradient is True}
                #if ?{traits.diagonal is True}
                    auto J = graphdot::tensor_view(gradient, nX, nJ);
                #else
                    auto J = graphdot::tensor_view(gradient, nX, nY, nJ);
                #endif
            #endif

            // solve the MLGK equation
            #if ?{traits.eval_gradient is True}
                solver_t::compute_duo(
                    node_kernel,
                    edge_kernel,
                    p_start,
                    g1, g2,
                    scratch,
                    shmem,
                    q, q0);
            #else
                solver_t::compute(
                    node_kernel,
                    edge_kernel,
                    g1, g2,
                    scratch,
                    shmem,
                    q, q0);
            #endif
            __syncthreads();

            // apply min-path truncation
            #if ?{traits.lmin == 1}
                for (int i = threadIdx.x; i < N; i += blockDim.x) {
                    int i1 = i / n2;
                    int i2 = i % n2;
                    scratch.x(i) -= node_kernel(g1.node[i1], g2.node[i2]) * q * q / (q0 * q0);
                }
            #endif
            __syncthreads();

            // write kernel matrix elements to output
            #if ?{traits.nodal == "block"}
                for (int i = threadIdx.x; i < N; i += blockDim.x) {
                    int i1 = i / n2;
                    int i2 = i % n2;
                    K(I1 + i1 + i2 * n1) =
                        scratch.x(i) * p_start(g1.node[i1]) * p_start(g2.node[i2]);
                }
            #elif ?{traits.nodal is True}
                #if ?{traits.diagonal is True}
                    for (int i1 = threadIdx.x; i1 < g1.n_node; i1 += blockDim.x) {
                        K(I1 + i1) = scratch.x(i1 + i1 * n1) * graphdot::ipow<2>(p_start(g1.node[i1]));
                    }
                #else
                    for (int i = threadIdx.x; i < N; i += blockDim.x) {
                        int i1 = i / n2;
                        int i2 = i % n2;
                        auto r = scratch.x(i) * p_start(g1.node[i1]) * p_start(g2.node[i2]);
                        K(I1 + i1, I2 + i2) = r;
                        #if ?{traits.symmetric is True}
                            if (job.x != job.y) K(I2 + i2, I1 + i1) = r;
                        #endif
                    }    
                #endif
            #elif ?{traits.nodal is False}
                // wipe output buffer for atomic accumulations
                if (threadIdx.x == 0) {
                    #if ?{traits.diagonal is True}
                        K(I1) = 0.f;
                    #else
                        K(I1, I2) = 0.f;
                        #if ?{traits.symmetric is True}
                            K(I2, I1) = 0.f;
                        #endif
                    #endif
                }

                __syncthreads();

                float32 sum = 0;
                for (int i = threadIdx.x; i < N; i += blockDim.x) {
                    int i1 = i / n2;
                    int i2 = i % n2;
                    sum += scratch.x(i) * p_start(g1.node[i1]) * p_start(g2.node[i2]);
                }
                sum = graphdot::cuda::warp_sum(sum);
                if (lane == 0) {
                    #if ?{traits.diagonal is True}
                        atomicAdd(K.at(I1), sum);
                    #else
                        atomicAdd(K.at(I1, I2), sum);
                        #if ?{traits.symmetric is True}
                            if (job.x != job.y) {
                                atomicAdd(K.at(I2, I1), sum);
                            }
                        #endif
                    #endif
                }
            #endif

            // optionally evaluate the gradient
            #if ?{traits.eval_gradient is True}

                #if ?{traits.nodal is True}
                    // auto jacobian
                #elif ?{traits.nodal is False}
                    auto jacobian = solver_t::derivative(
                        p_start,
                        node_kernel,
                        edge_kernel,
                        g1, g2,
                        scratch,
                        shmem,
                        q
                    );

                    #if ?{traits.diagonal is True}
                        for (int i = threadIdx.x; i < jacobian.size; i += blockDim.x) {
                            J(I1, i) = 0;
                        }
                        __syncthreads();
                        #pragma unroll (jacobian.size)
                        for(int i = 0; i < jacobian.size; ++i) {
                            auto j = graphdot::cuda::warp_sum(jacobian[i]);
                            if (lane == 0) {
                                atomicAdd(J.at(I1, i), j);
                            };
                        }
                    #else
                        for (int i = threadIdx.x; i < jacobian.size; i += blockDim.x) {
                            J(I1, I2, i) = 0;
                            #if ?{traits.symmetric is True}
                                if (job.x != job.y) J(I2, I1, i) = 0;
                            #endif
                        }
                        __syncthreads();
                        #pragma unroll (jacobian.size)
                        for(int i = 0; i < jacobian.size; ++i) {
                            auto j = graphdot::cuda::warp_sum(jacobian[i]);
                            if (lane == 0) {
                                atomicAdd(J.at(I1, I2, i), j);
                                #if ?{traits.symmetric is True}
                                    if (job.x != job.y) atomicAdd(J.at(I2, I1, i), j);
                                #endif
                            };
                        }
                    #endif
                #endif
                __syncthreads();
            #endif
        }
    }
}
