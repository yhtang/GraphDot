#include "hip/hip_runtime.h"
#include <array.h>
#include <basekernel.h>
#include <fmath.h>
#include <frozen_array.h>
#include <graph.h>
#include <marginalized_kernel.h>
#include <numpy_type.h>
#include <tensor_view.h>
#include <util_cuda.h>

using namespace graphdot::numpy_type;
using namespace graphdot::basekernel;
namespace solver_ns = graphdot::marginalized;

${node_kernel}
${edge_kernel}
${p_start}

using node_t = ${node_t};
using edge_t = ${edge_t};

using graph_t   = graphdot::graph_t<node_t, edge_t>;
using scratch_t = solver_ns::pcg_scratch_t;
using solver_t  = solver_ns::labeled_compact_block_dynsched_pcg<graph_t>;

__constant__ char shmem_bytes_per_warp[solver_t::shmem_bytes_per_warp];

extern "C" {
    __global__ void graph_kernel_solver(
        graph_t const   * graphs,
        scratch_t       * scratches,
        uint2           * jobs,
        uint            * starts,
        float32         * gramian,
        float32         * gradient,
        uint            * i_job_global,
        const uint        n_jobs,
        const uint        nX,
        const uint        nY,
        const uint        nJ,
        const float32     q,
        const float32     q0
    ) {
        extern __shared__ char shmem[];
        __shared__ uint i_job;

        const int lane = graphdot::cuda::laneid();
        auto scratch = scratches[blockIdx.x];

        while (true) {
            if (threadIdx.x == 0) i_job = atomicInc(i_job_global, 0xFFFFFFFF);
            __syncthreads();

            if (i_job >= n_jobs) break;

            const auto job = jobs[i_job];
            const auto g1  = graphs[job.x];
            const auto g2  = graphs[job.y];
            const auto I1  = starts[job.x];
            const auto I2  = starts[job.y];
            const uint n1  = g1.n_node;
            const uint n2  = g2.n_node;
            const uint N   = n1 * n2;

            if (?{traits.eval_gradient is True}) {
                solver_t::compute_duo(
                    node_kernel,
                    edge_kernel,
                    p_start,
                    g1, g2,
                    scratch,
                    shmem,
                    q, q0);
            } else {
                solver_t::compute(
                    node_kernel,
                    edge_kernel,
                    g1, g2,
                    scratch,
                    shmem,
                    q, q0);
            }
            __syncthreads();

            /********* post-processing *********/

            // apply starting probability and min-path truncation
            if (?{traits.lmin == 1}) {
                for (int i = threadIdx.x; i < N; i += blockDim.x) {
                    int i1 = i / n2;
                    int i2 = i % n2;
                    scratch.x(i) -= node_kernel(g1.node[i1], g2.node[i2]) * q * q / (q0 * q0);
                }
            }
            __syncthreads();

            // write to output buffer
            if (?{traits.nodal == "block"}) {
                for (int i = threadIdx.x; i < N; i += blockDim.x) {
                    int i1 = i / n2;
                    int i2 = i % n2;
                    gramian[I1 + i1 + i2 * n1] =
                        scratch.x(i) * p_start(g1.node[i1]) * p_start(g2.node[i2]);
                }
            }
            if (?{traits.nodal is True}) {
                if (?{traits.diagonal is True}) {
                    for (int i1 = threadIdx.x; i1 < g1.n_node; i1 += blockDim.x) {
                        gramian[I1 + i1] = scratch.x(i1 + i1 * n1) * graphdot::ipow<2>(p_start(g1.node[i1]));
                    }
                } else {
                    auto K = graphdot::tensor_view(gramian, nX, nY);
                    for (int i = threadIdx.x; i < N; i += blockDim.x) {
                        int i1 = i / n2;
                        int i2 = i % n2;
                        auto r = scratch.x(i) * p_start(g1.node[i1]) * p_start(g2.node[i2]);
                        K(I1 + i1, I2 + i2) = r;
                        if (?{traits.symmetric is True}) {
                            if (job.x != job.y) {
                                K(I2 + i2, I1 + i1) = r;
                            }
                        }
                    }    
                }
            }
            if (?{traits.nodal is False}) {
                // wipe output buffer for atomic accumulations
                if (threadIdx.x == 0) {
                    if (?{traits.diagonal is True}) {
                        gramian[I1] = 0.f;
                   } else {
                       gramian[I1 + I2 * nX] = 0.f;
                       if (?{traits.symmetric is True}) gramian[I2 + I1 * nX] = 0.f;
                   }   
                }

                __syncthreads();

                float32 sum = 0;
                for (int i = threadIdx.x; i < N; i += blockDim.x) {
                    int i1 = i / n2;
                    int i2 = i % n2;
                    sum += scratch.x(i) * p_start(g1.node[i1]) * p_start(g2.node[i2]);
                }
                sum = graphdot::cuda::warp_sum(sum);
                if (lane == 0) {
                    if (?{traits.diagonal is True}) {
                        atomicAdd(gramian + I1, sum);
                    } else {
                        auto K = graphdot::tensor_view(gramian, nX, nY);
                        atomicAdd(K.at(I1, I2), sum);
                        if (?{traits.symmetric is True}) {
                            if (job.x != job.y) {
                                atomicAdd(K.at(I2, I1), sum);
                            }
                        }
                    }
                }

                if (?{traits.eval_gradient is True}) {

                    auto jacobian = solver_t::derivative(
                        p_start,
                        node_kernel,
                        edge_kernel,
                        g1, g2,
                        scratch,
                        shmem,
                        q
                    );

                    // wipe output buffer for atomic accumulations
                    if (?{traits.diagonal is True}) {

                        auto J = graphdot::tensor_view(gradient, nX, nJ);

                        for (int i = threadIdx.x; i < jacobian.size; i += blockDim.x) {
                            J(I1, i) = 0;
                        }    

                        __syncthreads();

                        #pragma unroll (jacobian.size)
                        for(int i = 0; i < jacobian.size; ++i) {
                            auto j = graphdot::cuda::warp_sum(jacobian[i]);
                            if (lane == 0) {
                                atomicAdd(J.at(I1, i), j);
                            };
                        }
                    } else {

                        auto J = graphdot::tensor_view(gradient, nX, nY, nJ);

                        for (int i = threadIdx.x; i < jacobian.size; i += blockDim.x) {
                            J(I1, I2, i) = 0;
                            if (?{traits.symmetric is True} && job.x != job.y) {
                                J(I2, I1, i) = 0;
                            }
                        }

                        __syncthreads();

                        #pragma unroll (jacobian.size)
                        for(int i = 0; i < jacobian.size; ++i) {
                            auto j = graphdot::cuda::warp_sum(jacobian[i]);
                            if (lane == 0) {
                                atomicAdd(J.at(I1, I2, i), j);
                                if (?{traits.symmetric is True}) {
                                    if (job.x != job.y) {
                                        atomicAdd(J.at(I2, I1, i), j);
                                    }
                                }
                            };
                        }
                    }
                    __syncthreads();
                }
            }
        }
    }
}
